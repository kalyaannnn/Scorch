#include "hip/hip_runtime.h"
__host__ void cpu_to_cuda(Tensor* tensor)private{
    float* data_tmp;
    hipMalloc((void **)&data_tmp, tensor -> size * sizeof(float));
    hipMemcpy(data_temp, tensor -> data, tensor -> size * sizeof(float), hipMemcpyHostToDevice);

    tensor -> data = data_temp;
    tensor -> device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor -> device, device_str);

    printf("Successfully sent tensor to: %s \n", tensor -> device);
}

__host__ void cuda_to_cpu(Tensor* tensor){
    float* data_tmp = (float*)malloc(tensor -> size * sizeof(float));

    hipMemcpy(data_tmp, tensor -> data, tensor -> size * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(tensor -> data);

    tensor -> data = data_tmp;
    const char* device_str = "cpu";
    tensor -> device = (char*)malloc(strlen(device_str) + 1);
    strcpy(tensor -> device, device_str);

    printf("Succesfully sent tensor to %s \n", tensor -> device);
}

#define THREADS_PER_BLOCK 128

__global__ void add_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size){
        result_data[i] = data1[i] + data2[i];
    } 
}

__host__ void add_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data){
    int number_of_blocks = (tensor1 -> size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    add_tensor_cuda_kernel <<< number_of_blocks, THREADS_PER_BLOCK >>> (tensor1 -> data, tensor2 -> data, result_data, tensor1 -> size);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA Error : %s \n", hipGetErrorString(error));
        exit(-1);
    }
    hipDeviceSynchronize();
}

__global__ void sub_tensor_cuda_kernel(float* data1, float* data2, float* result_data, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size){
        result_data[i] = data1[i] - data2[i];
    }
}

__host__ void sub_tensor_cuda(Tensor* tensor1, Tensor* tensor2, float* result_data){
    int number_of_blocks = (tensor1 -> size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    sub_tensor_cuda_kernel <<< number_of_blocks, THREADS_PER_BLOCK >>> (tensor1 -> data, tensor2 -> data, result_data, tensor1 -> size);

    hipError_t error = hipGetLastError();
    if(error != hipSuccess){
        printf("CUDA Error : %s \n", hipGetErrorString(error));
        exit(-1);
    }
    hipDeviceSynchronize();
}